#include "hip/hip_runtime.h"
#include <stdio.h>
#include <inttypes.h>

#include "helper.cuh"
#include "framebuffer.cuh"
#include "cudaviewer.h"

#define RES_X 800
#define RES_Y 600

__global__ void fill_image(uint32_t* img, uint8_t b) {
    uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= RES_X || y >= RES_Y) return;

    uint8_t r = (uint8_t)(((float)x * 255.0) / (float)RES_X);
    uint8_t g = (uint8_t)(((float)y * 255.0) / (float)RES_Y);

    uint64_t index = x + y * RES_X;
    img[index] = (uint32_t)r + ((uint32_t)g << 8) + ((uint32_t)b << 16);
}

int main() {
    // Create a Vulkan shared framebuffer
    Framebuffer fb(make_int2(RES_X, RES_Y));

    // Initialize viewer
    hipDeviceProp_t deviceProps;
    checkCudaErrors(hipGetDeviceProperties(&deviceProps, 0));
    void* viewerCtx = viewer::init(fb.m_shareableHandle, fb.m_size, RES_X, RES_Y, (uint8_t*)&deviceProps.uuid.bytes);
    
    // Render main loop
    hipEvent_t event;
    checkCudaErrors(hipEventCreate(&event));
    
    uint8_t b = 0;
    while (true) {
        checkCudaErrors(hipEventRecord(event));
        fill_image<<<dim3(ceil(float(RES_X)/32.0), ceil(float(RES_Y)/32.0)), dim3(32, 32)>>>(fb.m_devPtr, b++);

        // Run the viewer's event loop while we wait to resubmit the CUDA kernel
        bool shouldClose = false;
        while (!(shouldClose = viewer::run_event_loop(viewerCtx)) && hipEventQuery(event) != hipSuccess);

        if (shouldClose) break;
    }

    checkCudaErrors(hipEventDestroy(event));

    // Deinit viewer
    viewer::deinit(viewerCtx);

    return EXIT_SUCCESS;
}
